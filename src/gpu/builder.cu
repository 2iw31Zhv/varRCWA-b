#include "builder.h"
#include "CudaLayerSampler.h"
#include "CudaFourierSolver2D.h"
#include <iostream>
#include <chrono>
#include <memory>
#include <utility>
#include <hipsolver.h>

NAMESPACE_BEGIN(acacia::gpu::em)
void build_rcwa_sample(int nCx, int nCy,
  const Real *x, 
  const Real *y,
  const std::complex<Real> *eps)
{
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);
  hipblasHandle_t cublasH = NULL;
  hipblasCreate(&cublasH);

  int nHx = 12;
  int nHy = 10;
  int nx = 2 * nHx + 1;
  int ny = 2 * nHy + 1;
  Real lambda = 1.55;

  Real *cu_x=nullptr, *cu_y=nullptr;
  complex_t *cu_eps=nullptr;
  hipMalloc(&cu_x, sizeof(Real)*nCx);
  hipMalloc(&cu_y, sizeof(Real)*nCy);
  hipMalloc(&cu_eps, sizeof(complex_t)*(nCx-1)*(nCy-1));

  hipMemcpy(cu_x, x, sizeof(Real)*nCx, hipMemcpyHostToDevice);
  hipMemcpy(cu_y, y, sizeof(Real)*nCy, hipMemcpyHostToDevice);
  hipMemcpy(cu_eps, eps, sizeof(complex_t)*(nCx-1)*(nCy-1),
    hipMemcpyHostToDevice);
  
  auto fourier = 
    std::make_shared<CudaFourierSolver2D>(cusolverH, nCx, nCy, cu_x, cu_y);

  CudaLayerSampler sampler(cublasH, cusolverH, fourier);

  
  size_t wSize = sampler.workspace_buffer_size(nx, ny);
  complex_t *workspace=nullptr;
  hipMalloc(&workspace, sizeof(complex_t)*wSize);
  size_t iSize = sampler.iworkspace_buffer_size(nx, ny);
  int *iworkspace=nullptr;
  hipMalloc(&iworkspace, sizeof(int)*iSize);

  int edgeSize = nx*ny;
  int blockSize = edgeSize * edgeSize;

  complex_t *Kx=nullptr, *Ky=nullptr;
  hipMalloc(&Kx, sizeof(complex_t)*blockSize);
  hipMalloc(&Ky, sizeof(complex_t)*blockSize);
  // TODO set and copy Kx, Ky from cpu

  complex_t *P00=nullptr, 
    *P01=nullptr, 
    *P10=nullptr, 
    *P11=nullptr;
  complex_t *Q00=nullptr,
    *Q01=nullptr,
    *Q10=nullptr,
    *Q11=nullptr;
  hipMalloc(&P00, sizeof(complex_t)*blockSize);
  hipMalloc(&P01, sizeof(complex_t)*blockSize);
  hipMalloc(&P10, sizeof(complex_t)*blockSize);
  hipMalloc(&P11, sizeof(complex_t)*blockSize);
  hipMalloc(&Q00, sizeof(complex_t)*blockSize);
  hipMalloc(&Q01, sizeof(complex_t)*blockSize);
  hipMalloc(&Q10, sizeof(complex_t)*blockSize);
  hipMalloc(&Q11, sizeof(complex_t)*blockSize);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  sampler.sample(cu_eps, Kx, Ky, lambda, nx, ny, 
    P00, P01, P10, P11,
    Q00, Q01, Q10, Q11,
    workspace, iworkspace);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop); 
  printf("loop time: %fms\n", milliseconds);
  
  hipFree(P00);
  hipFree(P01);
  hipFree(P10);
  hipFree(P11);
  hipFree(Q00);
  hipFree(Q01);
  hipFree(Q10);
  hipFree(Q11);

  hipFree(Kx);
  hipFree(Ky);

  hipFree(iworkspace);
  hipFree(workspace);
  hipFree(cu_x);
  hipFree(cu_y);
  hipFree(cu_eps);
  if (cublasH) hipblasDestroy(cublasH);
  if (cusolverH) hipsolverDnDestroy(cusolverH);
}

NAMESPACE_END(acacia::gpu::em)